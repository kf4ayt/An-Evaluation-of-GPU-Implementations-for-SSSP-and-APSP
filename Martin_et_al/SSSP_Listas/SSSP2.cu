

/*******

The code below is the original code, edited so that it would run on CUDA
Compute Capability 6.1 hardware (EVGA/NVIDIA GTX 1070) with CUDA v9.0.176.
The display driver being used is NVIDIA 384.111. The OS is Debian Linux v9
('Sid').

Charles W Johnson
April, 2018

*******/


///////////////////////////////////////
///////////////////////////////// SSSP2
///////////////////////////////////////


/* CWJ includes */

#include <hip/hip_runtime.h>

#include "comun.cu"

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }



#include <unistd.h>

#ifndef _SSSP2
#define _SSSP2

bool ejecutarIteracion_SSSP2 ( 
                       const unsigned int nVuelta, 
                       const dim3 grid, const dim3 threads, 
                       const unsigned int nv, const unsigned int na,
                       const unsigned int mem_size_V, const unsigned int mem_size_A, 
                       const unsigned int mem_size_C, const unsigned int mem_size_F,
                       const unsigned int infinito,   
                       bool* p_h, bool* f_h, unsigned int* c_h ,
                       const unsigned int* v_d, const unsigned int* a_d, const unsigned int* w_d, 
                       bool* p_d, bool* f_d, unsigned int* c_d)
{
    //RECUERDA: mem_size_V= (nv+1)*sizeof(unsigned int)

    // ACTUALIZANDO CAMINOS MINIMOS ESPECIALES: kernel1

    hipGetLastError(); // reset the runtime error variable to hipSuccess

    kernel1<<<grid,threads>>>(v_d, a_d, w_d, p_d, f_d, c_d);

    // check if kernel execution generated and error
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    hipDeviceSynchronize();

    //traer C de device a host
    copiarD2H((void*)c_h, (void*)c_d, mem_size_C); 

    //MINIMIZANDO LOS COSTES RECIEN ACTUALIZADOS
    unsigned int i;
    unsigned int minimo = infinito;
    
    for (i=0; i<nv; i++) {
        if (p_h[i] && (minimo>c_h[i])) {
            minimo = c_h[i];
        }
    }

    //ACTUALIZANDO LA FRONTERA: Kernel3

    //ejecutar último kernel 
    hipGetLastError(); // reset the runtime error variable to hipSuccess

    kernel3<<<grid,threads>>>( p_d, f_d, c_d, minimo);

    // check if kernel execution generated and error
    gpuErrchk(hipDeviceSynchronize());
    gpuErrchk(hipPeekAtLastError());

    hipDeviceSynchronize();

    //Llevar de device a la actualizacion de P
    copiarD2H((void*) p_h, (void*)p_d, mem_size_F);
  
    return (minimo==infinito);

}//ejecutarIteracion_SSSP2


void testGraph_SSSP2(const unsigned int nv, const unsigned int mem_size_V,
                     const unsigned int na, const unsigned int mem_size_A,
                     const unsigned int infinito,   
                     const unsigned int* v_h, const unsigned int* a_h, const unsigned int* w_h,
                     const unsigned int* reference)
{
    //RECUERDA: mem_size_V= (nv+1)*sizeof(unsigned int)

    unsigned int* v_d; //array de vértices device
    unsigned int* a_d; //array de aristas device
    unsigned int* w_d; //array de pesos device
   
    //copiar grafo de host a device
    inicializar_Grafo_Device(v_h, mem_size_V, v_d, 
                             a_h, mem_size_A, a_d,
                             w_h, w_d);

    unsigned int* c_h; //solución en el host
    unsigned int* c_d; //solución en el device
    unsigned int mem_size_C= mem_size_V-sizeof(unsigned int); //Descontar el tapon -4
    inicializar_Sol(c_h, c_d, nv, mem_size_C, infinito);
    
    bool* f_h; //frontera en el host
    bool* f_d; //frontera en el device
    unsigned int mem_size_F= sizeof(bool) * nv;
    inicializar_Frontera(f_h, f_d, nv, mem_size_F);

    bool* p_h; //pendientes por procesar 
    bool* p_d; //pendientes por procesar 
    inicializar_Pendientes(p_h, p_d, nv, mem_size_F);

#ifdef DEBUG
    //DEPURACION
    printf("\nnv= %i\n", nv);
    printf("na= %i\n", na);
    printf("mem_size_V= %i\n", mem_size_V);
    printf("mem_size_A= %i\n", mem_size_A);
    printf("mem_size_F= %i\n\n", mem_size_F);
#endif // _DEBUG

    // setup execution parameters
    unsigned int num_threadsInBlock= NUM_THREADS_IN_BLOCK;
    //unsigned int num_blocksInGrid= nv/num_threadsInBlock;     // original code but the next line is better
    unsigned int num_blocksInGrid = (nv + (num_threadsInBlock-1)) / num_threadsInBlock;

    dim3  grid(num_blocksInGrid, 1, 1);
    dim3  threads(num_threadsInBlock, 1, 1);

    /* Updated timer code for CUDA 9 */

    hipEvent_t timerStart, timerStop;
    float time;

    //EJECUTAR VUELTAS  
    bool ultima= false;
    unsigned int i= 0;

    // start things
    hipEventCreate(&timerStart);
    hipEventCreate(&timerStop);
    hipEventRecord(timerStart, 0);

    while(!ultima){
        i++;
        ultima= ejecutarIteracion_SSSP2( i, 
                                         grid, threads, 
                                         nv, na,
                                         mem_size_V, mem_size_A, mem_size_C, mem_size_F,
                                         infinito, 
                                         p_h, f_h, c_h, 
                                         v_d, a_d, w_d, p_d, f_d, c_d);
    }//while

    hipEventRecord(timerStop, 0);
    hipEventSynchronize(timerStop);

    hipEventElapsedTime(&time, timerStart, timerStop);
    hipEventDestroy(timerStart);
    hipEventDestroy(timerStop);
    //printf("%.6f", time);
    printf("Runtime for SSSP2 algorithm is: %.6f ms\n", time);

    copiarD2H((void*)c_h, (void*)c_d, mem_size_C); 

    // cleanup memory
    free(f_h);
    free(p_h);

    hipFree(v_d);
    hipFree(a_d);
    hipFree(w_d);

    hipFree(c_d);
    hipFree(f_d);
    hipFree(p_d);

    // check result
    //CUTBoolean res = cutComparei( (int*)reference, (int*)c_h, nv);
    //printf( "%s\t", (1 == res) ? "OK" : "FAILED");

    //mostrarUI(c_h, nv, "c_h");
    //mostrarUI(reference, nv, "reference");


    // cleanup memory
    free(c_h);

}

#endif //#ifndef _SSSP2


